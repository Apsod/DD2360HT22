#include "hip/hip_runtime.h"

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <random>
#include <vector>

#define DataType double
#ifndef STREAMS
#define STREAMS 4 
#endif

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < len)
    out[i] = in1[i] + in2[i];
}


//@@ Insert code to implement timer start

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

//@@ Insert code to implement timer stop

inline __host__ __device__ int divUp(int denom, int numer){
  return (denom + numer - 1) / numer;
}

inline __host__ __device__ int asMultipleOf(int value, int factor) {
  // Returns the smallest value larger or equal to 
  // "value" that is a multiple of "factor"
  return divUp(value, factor) * factor;
}


int main(int argc, char **argv) {
  
  int inputLength;
  int segment_size;
  float elapsed; 
  DataType *hostMemory;

  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;

  DataType *resultRef;

  DataType *deviceMemory;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  
  // Initialize the 1D grid and block dimensions here

  // Read in inputLength from args
  inputLength = atoi(argv[1]);
  segment_size = atoi(argv[2]);

  segment_size = min(segment_size, inputLength); 


  int segments = divUp(inputLength, segment_size);
  int BLOCKS = divUp(segment_size, 1024);
  int TPB = divUp(segment_size, BLOCKS);

  printf("The input length is   : %d\n", inputLength);
  printf("number of streams     : %d\n", STREAMS);
  printf("number of segments    : %d\n", segments);
  printf("number of blocks/str  : %d\n", BLOCKS);
  printf("number of threads/str : %d\n", TPB);
  
  hipEvent_t start;
  hipEvent_t stop; 

  checkCuda(hipEventCreate(&start));
  checkCuda(hipEventCreate(&stop));

  // Allocate Host memory for input and output
  //checkCuda(hipHostMalloc(&hostMemory, inputLength * 3 * sizeof *hostMemory));
  checkCuda(hipHostAlloc(&hostMemory, inputLength * 3 * sizeof(DataType), hipHostMallocDefault));

  hostInput1 = &hostMemory[0*inputLength];
  hostInput2 = &hostMemory[1*inputLength];
  hostOutput = &hostMemory[2*inputLength];


  resultRef = (DataType*) malloc(inputLength * sizeof *resultRef);
  
  
  // Initialize hostInput1 and hostInput2 to random numbers.

  std::normal_distribution<DataType> distribution(0.0, 10.0);
  std::default_random_engine gen(1337);
  
  for (DataType *ptr : {hostInput1, hostInput2}) {
    for (int i=0; i<inputLength; ++i){
      ptr[i] = distribution(gen);
    }
  }
  
  hipEventRecord(start);
  // Create reference result in CPU
  for (int i=0; i < inputLength; ++i)
    resultRef[i] = hostInput1[i] + hostInput2[i];
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop); 
  printf("host execution time   (ms) : %f\n", elapsed);

  hipStream_t streams[STREAMS];
  for (int i=0; i < STREAMS; ++i)
    hipStreamCreate(&streams[i]);
  // Allocate GPU memory
  
  hipEventRecord(start);
  checkCuda(hipMalloc(&deviceMemory, inputLength * 3 * sizeof(DataType)));
  deviceInput1 = &deviceMemory[0*inputLength];
  deviceInput2 = &deviceMemory[1*inputLength];
  deviceOutput = &deviceMemory[2*inputLength];
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop); 
  printf("cudamalloc            (ms) : %f\n", elapsed);

  // Copy memory to the GPU
  


  // Launch the streams
  hipEventRecord(start);
  for (int i=0; i < segments; ++i){
    int offset = i * segment_size;
    int len = min(segment_size, inputLength - offset);
    int bytes = len * sizeof(DataType); 
    hipStream_t stream = streams[i % STREAMS];
    hipMemcpyAsync(&deviceInput1[offset], &hostInput1[offset], bytes,
        hipMemcpyHostToDevice, stream
        ); 
    hipMemcpyAsync(&deviceInput2[offset], &hostInput2[offset], bytes,
        hipMemcpyHostToDevice, stream
        ); 
    vecAdd<<<BLOCKS, TPB, 0, stream>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], len);
    hipMemcpyAsync(&hostOutput[offset], &deviceOutput[offset], bytes,
        hipMemcpyDeviceToHost, stream
        ); 
  }
  hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop); 
  printf("kernel execution time (ms) : %f\n", elapsed);
  printf("GFLOP/s                    : %f\n", ((float) inputLength) / (elapsed * 1e6));

  DataType m = hostOutput[0] - resultRef[0];
  DataType m2 = 0.0;

  DataType maximumError = 0.0;

  int errs = 0;

  for (int i=1; i<inputLength; ++i) {
    DataType delta;
    DataType weight;
    DataType diff;

    weight = DataType(i);
    diff = hostOutput[i] - resultRef[i];

    errs += diff == 0.0 ? 0: 1; 

    delta = diff - m;

    m += delta / (weight + 1.0);
    m2 += pow(delta, 2) * weight / (weight + 1.0);
    maximumError = max(abs(diff), maximumError);
  }

  m2 = sqrt(m2 / DataType(inputLength-1));

  printf("mean difference       : %f\n", m);
  printf("standard deviation    : %f\n", m2);
  printf("maximum error         : %f\n", maximumError);
  printf("number of errors      : %d\n", errs);
  
  // Free the GPU memory

  hipEventDestroy(start);
  hipEventDestroy(stop);

  for (int i=0; i < STREAMS; ++i)
    hipStreamDestroy(streams[i]);

  hipFree(deviceMemory);

  hipHostFree(hostMemory);

  return 0;
}
