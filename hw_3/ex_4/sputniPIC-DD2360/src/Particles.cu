#include "hip/hip_runtime.h"
#include "Particles.h"
#include "Alloc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/** allocate particle arrays */
void particle_allocate(struct parameters* param, struct particles* part, int is)
{
    
    // set species ID
    part->species_ID = is;
    // number of particles
    part->nop = param->np[is];
    // maximum number of particles
    part->npmax = param->npMax[is];
    
    // choose a different number of mover iterations for ions and electrons
    if (param->qom[is] < 0){  //electrons
        part->NiterMover = param->NiterMover;
        part->n_sub_cycles = param->n_sub_cycles;
    } else {                  // ions: only one iteration
        part->NiterMover = 1;
        part->n_sub_cycles = 1;
    }
    
    // particles per cell
    part->npcelx = param->npcelx[is];
    part->npcely = param->npcely[is];
    part->npcelz = param->npcelz[is];
    part->npcel = part->npcelx*part->npcely*part->npcelz;
    
    // cast it to required precision
    part->qom = (FPpart) param->qom[is];
    
    long npmax = part->npmax;
    
    // initialize drift and thermal velocities
    // drift
    part->u0 = (FPpart) param->u0[is];
    part->v0 = (FPpart) param->v0[is];
    part->w0 = (FPpart) param->w0[is];
    // thermal
    part->uth = (FPpart) param->uth[is];
    part->vth = (FPpart) param->vth[is];
    part->wth = (FPpart) param->wth[is];
    
    
    //////////////////////////////
    /// ALLOCATION PARTICLE ARRAYS
    //////////////////////////////
    part->x = new FPpart[npmax];
    part->y = new FPpart[npmax];
    part->z = new FPpart[npmax];
    // allocate velocity
    part->u = new FPpart[npmax];
    part->v = new FPpart[npmax];
    part->w = new FPpart[npmax];
    // allocate charge = q * statistical weight
    part->q = new FPinterp[npmax];
    
}
/** deallocate */
void particle_deallocate(struct particles* part)
{
    // deallocate particle variables
    delete[] part->x;
    delete[] part->y;
    delete[] part->z;
    delete[] part->u;
    delete[] part->v;
    delete[] part->w;
    delete[] part->q;
}


void inner_loop(struct particles* part, struct EMfield* field, struct grid* grd, struct parameters* param, int pix){
    // auxiliary variables
    FPpart dt_sub_cycling = (FPpart) param->dt/((double) part->n_sub_cycles);
    FPpart dto2 = .5*dt_sub_cycling, qomdt2 = part->qom*dto2/param->c;
    FPpart omdtsq, denom;
    FPpart3 vt; 
    int3 i; 
    int3 fi;

    char3 peroidic = make_char3(param->PERIODICX, param->PERIODICY, param->PERIODICZ); 

    FPfield3 E, B;

    // local (to the particle) electric and magnetic field
    FPfield3 El, Bl;
    
    // interpolation densities
    FPfield weight; 
    FPfield3 N[2]; 

    // intermediate particle position and velocity
    FPpart3 p_;
    FPpart3 v_;
    
    FPpart3 p;
    FPpart3 v; 
    
    p_ = make_fppart3(part->x[pix], part->y[pix], part->z[pix]);

    p = p_;

    v = make_fppart3(part->u[pix], part->v[pix], part->w[pix]);

    double3 start = make_double3(grd->xStart, grd->yStart, grd->zStart);
    double3 L = make_double3(grd->Lx, grd->Ly, grd->Lz);
    FPfield3 invd = make_fpfield3(grd->invdx, grd->invdy, grd->invdz); 
    FPfield invVOL = grd->invVOL;

    // calculate the average velocity iteratively
    // THIS LOOP IS SEQUENTIAL
    for(int innter=0; innter < part->NiterMover; innter++){
        // interpolation G-->P
        i = 2 + make_int3((p - make_float3(start)) * invd);

        // calculate weights
        N[0] = make_fpfield3(
                p.x - grd->XN[i.x - 1][i.y][i.z],
                p.y - grd->YN[i.x][i.y - 1][i.z],
                p.z - grd->ZN[i.x][i.y][i.z - 1]
        );
        N[1] = make_fpfield3(
                grd->XN[i.x][i.y][i.z] - p.x,
                grd->YN[i.x][i.y][i.z] - p.y,
                grd->ZN[i.x][i.y][i.z] - p.z
        );

        // set to zero local electric and magnetic field
        El = make_fpfield3(0.0,0.0,0.0); 
        Bl = make_fpfield3(0.0,0.0,0.0); 
        
        // THIS LOOP IS PARALLELIZABLE (but only 8 * 6 = 48 operations unrolled)
        for (int ii=0; ii < 2; ii++)
            for (int jj=0; jj < 2; jj++)
                for(int kk=0; kk < 2; kk++){
                    weight = N[ii].x * N[jj].y * N[kk].z * invVOL;
                    fi = i - make_int3(ii, jj, kk);
                    E = make_fpfield3(
                            field->Ex[fi.x][fi.y][fi.z],
                            field->Ey[fi.x][fi.y][fi.z],
                            field->Ez[fi.x][fi.y][fi.z]
                            );
                    B = make_fpfield3(
                            field->Bxn[fi.x][fi.y][fi.z],
                            field->Byn[fi.x][fi.y][fi.z],
                            field->Bzn[fi.x][fi.y][fi.z]
                            );
                    El += weight * E;
                    Bl += weight * B; 
                }
        
        // end interpolation
        omdtsq = qomdt2*qomdt2*(dot(Bl, Bl));
        denom = 1.0/(1.0 + omdtsq);
        // solve the position equation
        vt = v + qomdt2*El;
        // solve the velocity equation
        v_ = (vt + qomdt2 * (cross(vt, Bl) + qomdt2 * dot(vt, Bl) * Bl)) * denom;
        // update position
        p = p_ + v_ * dto2; 
    } // end of iteration
    
    v = 2.0 * v_ - v; 
    p = p_ + v_ * dt_sub_cycling;

    if (p.x > L.x){
        if (param->PERIODICX==true){ // PERIODIC
            p.x = p.x - L.x;
        } else { // REFLECTING BC
            v.x = -v.x;
            p.x = 2*L.x - p.x;
        }
    }
                                                                
    if (p.x < 0){
        if (param->PERIODICX==true){ // PERIODIC
           p.x = p.x + L.x;
        } else { // REFLECTING BC
            v.x = -v.x;
            p.x = -p.x;
        }
    }

    if (p.y > L.y){
        if (param->PERIODICY==true){ // PERIODIC
            p.y = p.y - L.y;
        } else { // REFLECTING BC
            v.y = -v.y;
            p.y = 2*L.y - p.y;
        }
    }
                                                                
    if (p.y < 0){
        if (param->PERIODICY==true){ // PERIODIC
           p.y = p.y + L.y;
        } else { // REFLECTING BC
            v.y = -v.y;
            p.y = -p.y;
        }
    }

    if (p.z > L.z){
        if (param->PERIODICZ==true){ // PERIODIC
            p.z = p.z - L.z;
        } else { // REFLECTING BC
            v.z = -v.z;
            p.z = 2*L.z - p.z;
        }
    }
                                                                
    if (p.z < 0){
        if (param->PERIODICZ==true){ // PERIODIC
           p.z = p.z + L.z;
        } else { // REFLECTING BC
            v.z = -v.z;
            p.z = -p.z;
        }
    }

    part->u[pix] = v.x; 
    part->v[pix] = v.y; 
    part->w[pix] = v.z; 

    part->x[pix] = p.x; 
    part->y[pix] = p.y; 
    part->z[pix] = p.z; 
}
                                                                        

/** particle mover */
int mover_PC(struct particles* part, struct EMfield* field, struct grid* grd, struct parameters* param)
{
    // print species and subcycling
    std::cout << "***  MOVER with SUBCYCLYING "<< param->n_sub_cycles << " - species " << part->species_ID << " ***" << std::endl;

    // start subcycling
    for (int i_sub=0; i_sub <  part->n_sub_cycles; i_sub++){
        // move each particle with new fields
        for (int i=0; i <  part->nop; i++){
            inner_loop(part, field, grd, param, i);
        }  // end of subcycling
    } // end of one particle

    return(0); // exit succcesfully
} // end of the mover

/** Interpolation Particle --> Grid: This is for species */
void interpP2G(struct particles* part, struct interpDensSpecies* ids, struct grid* grd)
{
    
    // arrays needed for interpolation
    FPpart weight[2][2][2];
    FPpart temp[2][2][2];
    FPpart xi[2], eta[2], zeta[2];
    
    // index of the cell
    int ix, iy, iz;
    
    
    for (register long long i = 0; i < part->nop; i++) {
        
        // determine cell: can we change to int()? is it faster?
        ix = 2 + int (floor((part->x[i] - grd->xStart) * grd->invdx));
        iy = 2 + int (floor((part->y[i] - grd->yStart) * grd->invdy));
        iz = 2 + int (floor((part->z[i] - grd->zStart) * grd->invdz));
        
        // distances from node
        xi[0]   = part->x[i] - grd->XN[ix - 1][iy][iz];
        eta[0]  = part->y[i] - grd->YN[ix][iy - 1][iz];
        zeta[0] = part->z[i] - grd->ZN[ix][iy][iz - 1];
        xi[1]   = grd->XN[ix][iy][iz] - part->x[i];
        eta[1]  = grd->YN[ix][iy][iz] - part->y[i];
        zeta[1] = grd->ZN[ix][iy][iz] - part->z[i];
        
        // calculate the weights for different nodes
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    weight[ii][jj][kk] = part->q[i] * xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;
        
        //////////////////////////
        // add charge density
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->rhon[ix - ii][iy - jj][iz - kk] += weight[ii][jj][kk] * grd->invVOL;
        
        
        ////////////////////////////
        // add current density - Jx
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[i] * weight[ii][jj][kk];
        
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->Jx[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        ////////////////////////////
        // add current density - Jy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->v[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->Jy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        
        ////////////////////////////
        // add current density - Jz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->w[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->Jz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        ////////////////////////////
        // add pressure pxx
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[i] * part->u[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pxx[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        ////////////////////////////
        // add pressure pxy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[i] * part->v[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pxy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        
        /////////////////////////////
        // add pressure pxz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[i] * part->w[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pxz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        /////////////////////////////
        // add pressure pyy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->v[i] * part->v[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pyy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        /////////////////////////////
        // add pressure pyz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->v[i] * part->w[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pyz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        /////////////////////////////
        // add pressure pzz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->w[i] * part->w[i] * weight[ii][jj][kk];
        for (int ii=0; ii < 2; ii++)
            for (int jj=0; jj < 2; jj++)
                for(int kk=0; kk < 2; kk++)
                    ids->pzz[ix -ii][iy -jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
    
    }
   
}

