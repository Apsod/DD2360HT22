#include "hip/hip_runtime.h"
#include "Particles.h"
#include "Alloc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/** allocate particle arrays */
void particle_allocate(struct parameters* param, struct particles* part, int is)
{
    
    // set species ID
    part->species_ID = is;
    // number of particles
    part->nop = param->np[is];
    // maximum number of particles
    part->npmax = param->npMax[is];
    
    // choose a different number of mover iterations for ions and electrons
    if (param->qom[is] < 0){  //electrons
        part->NiterMover = param->NiterMover;
        part->n_sub_cycles = param->n_sub_cycles;
    } else {                  // ions: only one iteration
        part->NiterMover = 1;
        part->n_sub_cycles = 1;
    }
    
    // particles per cell
    part->npcelx = param->npcelx[is];
    part->npcely = param->npcely[is];
    part->npcelz = param->npcelz[is];
    part->npcel = part->npcelx*part->npcely*part->npcelz;
    
    // cast it to required precision
    part->qom = (FPpart) param->qom[is];
    
    long npmax = part->npmax;
    
    // initialize drift and thermal velocities
    // drift
    part->u0 = (FPpart) param->u0[is];
    part->v0 = (FPpart) param->v0[is];
    part->w0 = (FPpart) param->w0[is];
    // thermal
    part->uth = (FPpart) param->uth[is];
    part->vth = (FPpart) param->vth[is];
    part->wth = (FPpart) param->wth[is];
    
    
    //////////////////////////////
    /// ALLOCATION PARTICLE ARRAYS
    //////////////////////////////
    part->x = new FPpart[npmax];
    part->y = new FPpart[npmax];
    part->z = new FPpart[npmax];
    // allocate velocity
    part->u = new FPpart[npmax];
    part->v = new FPpart[npmax];
    part->w = new FPpart[npmax];
    // allocate charge = q * statistical weight
    part->q = new FPinterp[npmax];
    
}
/** deallocate */
void particle_deallocate(struct particles* part)
{
    // deallocate particle variables
    delete[] part->x;
    delete[] part->y;
    delete[] part->z;
    delete[] part->u;
    delete[] part->v;
    delete[] part->w;
    delete[] part->q;
}

int get_size(int3 stride){
    return stride.x * stride.y * stride.z;
}

int get_ptr(int3 i, int3 stride){
    return i.x * stride.y * stride.z + i.y * stride.z + i.z;
}


void prep_grid(struct grid* grd, FPfield3* out)
{
    int ptr; 
    int X = grd->nxn; 
    int Y = grd->nyn; 
    int Z = grd->nzn; 
    for(int x=1; x<X; x++){
        for(int y=1; y<Y; y++){
            for(int z=1; z<Z; z++){
                ptr = 2 * (x * Y * Z + y * Z + z);
                out[ptr] = make_fpfield3(
                        grd->XN[x - 1][y][z],
                        grd->YN[x][y - 1][z],
                        grd->ZN[x][y][z - 1]
                        );
                out[ptr+1] = make_fpfield3(
                        grd->XN[x][y][z],
                        grd->YN[x][y][z],
                        grd->ZN[x][y][z]
                    );
            }
        }
    }
}

void inner_loop(
        //struct particles* part,
        FPpart3* pos,
        FPpart3* vel,
        struct EMfield* field,
        FPpart3* grid,
        int3 grid_stride,
        //struct grid* grd,
        double3 L,
        FPfield3 invd,
        FPfield invVOL,
        FPpart dt_sub_cycling,
        FPpart qomdt2,
        char3 periodic,
        int NiterMover,
        int pix)
{
    // auxiliary variables
    FPpart dto2 = .5*dt_sub_cycling;
    FPpart omdtsq, denom;
    FPpart3 vt; 
    int3 i; 
    int3 fi;
    int ptr; 

    FPfield3 E, B;

    // local (to the particle) electric and magnetic field
    FPfield3 El, Bl;
    
    // interpolation densities
    FPfield weight; 
    FPfield3 N[2]; 

    // intermediate particle position and velocity
    FPpart3 p_;
    FPpart3 v_;
    
    FPpart3 p;
    FPpart3 v; 
    
    p_ = pos[pix];


    p = p_;

    v = vel[pix];

    
    // start is always zero, and the original code can't handle wrapping when it isnt.
    //double3 start = make_double3(grd->xStart, grd->yStart, grd->zStart);

    // calculate the average velocity iteratively
    // THIS LOOP IS SEQUENTIAL
    for(int innter=0; innter < NiterMover; innter++){
        // interpolation G-->P
        i = 2 + make_int3(p * invd);

        // calculate weights
        ptr = get_ptr(i, grid_stride)*2;
        N[0] = p - grid[ptr];
        N[1] = grid[ptr + 1] - p;

        // set to zero local electric and magnetic field
        El = make_fpfield3(0.0,0.0,0.0); 
        Bl = make_fpfield3(0.0,0.0,0.0); 
        
        // THIS LOOP IS PARALLELIZABLE (but only 8 * 6 = 48 operations unrolled)
        for (int ii=0; ii < 2; ii++)
            for (int jj=0; jj < 2; jj++)
                for(int kk=0; kk < 2; kk++){
                    weight = N[ii].x * N[jj].y * N[kk].z * invVOL;
                    fi = i - make_int3(ii, jj, kk);
                    E = make_fpfield3(
                            field->Ex[fi.x][fi.y][fi.z],
                            field->Ey[fi.x][fi.y][fi.z],
                            field->Ez[fi.x][fi.y][fi.z]
                            );
                    B = make_fpfield3(
                            field->Bxn[fi.x][fi.y][fi.z],
                            field->Byn[fi.x][fi.y][fi.z],
                            field->Bzn[fi.x][fi.y][fi.z]
                            );
                    El += weight * E;
                    Bl += weight * B; 
                }
        
        // end interpolation
        omdtsq = qomdt2*qomdt2*(dot(Bl, Bl));
        denom = 1.0/(1.0 + omdtsq);
        // solve the position equation
        vt = v + qomdt2*El;
        // solve the velocity equation
        v_ = (vt + qomdt2 * (cross(vt, Bl) + qomdt2 * dot(vt, Bl) * Bl)) * denom;
        // update position
        p = p_ + v_ * dto2; 
    } // end of iteration
    
    v = 2.0 * v_ - v; 
    p = p_ + v_ * dt_sub_cycling;

    if (p.x > L.x){
        if (periodic.x){ // PERIODIC
            p.x = p.x - L.x;
        } else { // REFLECTING BC
            v.x = -v.x;
            p.x = 2*L.x - p.x;
        }
    }
                                                                
    if (p.x < 0){
        if (periodic.x){ // PERIODIC
           p.x = p.x + L.x;
        } else { // REFLECTING BC
            v.x = -v.x;
            p.x = -p.x;
        }
    }

    if (p.y > L.y){
        if (periodic.y){ // PERIODIC
            p.y = p.y - L.y;
        } else { // REFLECTING BC
            v.y = -v.y;
            p.y = 2*L.y - p.y;
        }
    }
                                                                
    if (p.y < 0){
        if (periodic.y){ // PERIODIC
           p.y = p.y + L.y;
        } else { // REFLECTING BC
            v.y = -v.y;
            p.y = -p.y;
        }
    }

    if (p.z > L.z){
        if (periodic.z){ // PERIODIC
            p.z = p.z - L.z;
        } else { // REFLECTING BC
            v.z = -v.z;
            p.z = 2*L.z - p.z;
        }
    }
                                                                
    if (p.z < 0){
        if (periodic.z){ // PERIODIC
           p.z = p.z + L.z;
        } else { // REFLECTING BC
            v.z = -v.z;
            p.z = -p.z;
        }
    }
    vel[pix] = v;
    pos[pix] = p; 
}


/** particle mover */
int mover_PC(struct particles* part, struct EMfield* field, struct grid* grd, struct parameters* param)
{
    // print species and subcycling
    std::cout << "***  MOVER with SUBCYCLYING "<< param->n_sub_cycles << " - species " << part->species_ID << " ***" << std::endl;

    FPpart3 *pos, *vel, *grid;

    pos = new FPpart3[part->nop]; 
    vel = new FPpart3[part->nop];
    int3 grid_stride = make_int3(grd->nxn, grd->nyn, grd->nzn);
    // std::cout << "*** ALLOCATING GRID. SIZE: " << get_size(grid_stride) * 2 * 4 * 3 << " Bytes.  ***" << std::endl; 
    grid = new FPpart3[get_size(grid_stride) * 2];

    // std::cout << "*** PREPPING GRID ***" << std::endl; 
    prep_grid(grd, grid); 

    FPpart dt_sub_cycling = (FPpart) param->dt/((double) part->n_sub_cycles);
    FPpart dto2 = .5 * dt_sub_cycling;
    FPpart qomdt2 = part->qom*dto2/param->c;
    char3 periodic = make_char3(param->PERIODICX, param->PERIODICY, param->PERIODICZ);
    int NiterMover = part->NiterMover;

    double3 L = make_double3(grd->Lx, grd->Ly, grd->Lz);
    FPfield3 invd = make_fpfield3(grd->invdx, grd->invdy, grd->invdz); 
    FPfield invVOL = grd->invVOL;


    // std::cout << "*** PREPPING POS & VEL ***" << std::endl; 
    for (int i=0; i<part->nop; ++i){
        pos[i] = make_fppart3(part->x[i], part->y[i], part->z[i]);
        vel[i] = make_fppart3(part->u[i], part->v[i], part->w[i]);
    }

    // std::cout << "*** STARTING ***" << std::endl; 
    // start subcycling
    for (int i_sub=0; i_sub <  part->n_sub_cycles; i_sub++){
        // move each particle with new fields
        for (int i=0; i <  part->nop; i++){
            inner_loop(pos, vel, field, grid, grid_stride, L, invd, invVOL, dt_sub_cycling, qomdt2, periodic, NiterMover, i);
        }  // end of subcycling
    } // end of one particle

    for (int i=0; i<part->nop; ++i){
        part->x[i] = pos[i].x;
        part->y[i] = pos[i].y;
        part->z[i] = pos[i].z;
        part->u[i] = vel[i].x;
        part->v[i] = vel[i].y;
        part->w[i] = vel[i].z;
    }

    return(0); // exit succcesfully
} // end of the mover

/** Interpolation Particle --> Grid: This is for species */
void interpP2G(struct particles* part, struct interpDensSpecies* ids, struct grid* grd)
{
    
    // arrays needed for interpolation
    FPpart weight[2][2][2];
    FPpart temp[2][2][2];
    FPpart xi[2], eta[2], zeta[2];
    
    // index of the cell
    int ix, iy, iz;
    
    
    for (register long long i = 0; i < part->nop; i++) {
        
        // determine cell: can we change to int()? is it faster?
        ix = 2 + int (floor((part->x[i] - grd->xStart) * grd->invdx));
        iy = 2 + int (floor((part->y[i] - grd->yStart) * grd->invdy));
        iz = 2 + int (floor((part->z[i] - grd->zStart) * grd->invdz));
        
        // distances from node
        xi[0]   = part->x[i] - grd->XN[ix - 1][iy][iz];
        eta[0]  = part->y[i] - grd->YN[ix][iy - 1][iz];
        zeta[0] = part->z[i] - grd->ZN[ix][iy][iz - 1];
        xi[1]   = grd->XN[ix][iy][iz] - part->x[i];
        eta[1]  = grd->YN[ix][iy][iz] - part->y[i];
        zeta[1] = grd->ZN[ix][iy][iz] - part->z[i];
        
        // calculate the weights for different nodes
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    weight[ii][jj][kk] = part->q[i] * xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;
        
        //////////////////////////
        // add charge density
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->rhon[ix - ii][iy - jj][iz - kk] += weight[ii][jj][kk] * grd->invVOL;
        
        
        ////////////////////////////
        // add current density - Jx
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[i] * weight[ii][jj][kk];
        
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->Jx[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        ////////////////////////////
        // add current density - Jy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->v[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->Jy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        
        ////////////////////////////
        // add current density - Jz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->w[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->Jz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        ////////////////////////////
        // add pressure pxx
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[i] * part->u[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pxx[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        ////////////////////////////
        // add pressure pxy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[i] * part->v[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pxy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        
        /////////////////////////////
        // add pressure pxz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[i] * part->w[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pxz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        /////////////////////////////
        // add pressure pyy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->v[i] * part->v[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pyy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        /////////////////////////////
        // add pressure pyz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->v[i] * part->w[i] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pyz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
        
        
        /////////////////////////////
        // add pressure pzz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->w[i] * part->w[i] * weight[ii][jj][kk];
        for (int ii=0; ii < 2; ii++)
            for (int jj=0; jj < 2; jj++)
                for(int kk=0; kk < 2; kk++)
                    ids->pzz[ix -ii][iy -jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
    
    }
   
}

